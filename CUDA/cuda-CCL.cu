#include "hip/hip_runtime.h"
/****************************************************************************
 *
 ****************************************************************************/

#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

#define BLKDIM 32

typedef struct{
	int h,w;
	int connectivity;
	int *image;
	int *result;
}bitmap;

/**
 * Reads the input bitmap.
 */
void read_input( FILE *f, bitmap *bm )
{
    int i, con, h, w, pixel;
	int size;
    int *pixels;
	int *positions;
    
    if ( 1 != fscanf(f, "%d", &con) ) {
        fprintf(stderr, "Error: cannot read connectivity\n");
        exit(EXIT_FAILURE);
    }
    if ( con != 4 && con != 8 ) {
        fprintf(stderr, "Error: connectivity cannot be %d. The only acceptable values are 4 and 8\n", con);
        exit(EXIT_FAILURE);
    }
    bm->connectivity = con;
    if ( 2 != fscanf(f, "%d %d", &h, &w) ){
		fprintf(stderr, "Error: cannot to read image sizes\n");
		exit(EXIT_FAILURE);
	}
    assert(h > 0);
    assert(w > 0);
    bm->h = h;
    bm->w = w;

	size = h * w;

    pixels = (int*)malloc( size * sizeof(int) );
	positions = (int*)malloc( size * sizeof(int) );
    assert(pixels);
	assert(positions);

    for (i=0; i<size; i++) {
		if (1 != fscanf(f, "%d ", &pixel)) {
			fprintf(stderr, "Error: cannot read the value of pixel at %d, %d\n", i, size % i);
			exit(EXIT_FAILURE);
		}
		assert(pixel == 0 || pixel == 1);
		pixels[i] = pixel;
		positions[i] = i;
    }
    bm->image = pixels;
    bm->result = positions;
}


/**
 * Reads the bitmap and creates the equivalences found in rows 
 */
__global__ void row_equivalences( int *input, int *res, int w, int h ){

    int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	//The position is increased by one, so the position 0 is reserved to the background value and it's easier to access the other values later.
    int pos = (j + i * w) + 1;
	int left = (j - 1) + i * w;

	if ( i == 0 && j == 0 ){
		res[0] = 0;
	}

	if ( i < h && j < w ){
    	if ( j == 0 ){
        	res[pos] = pos * input[pos - 1];
		} else {
			if ( input[pos - 1] ){
				if ( input[left] ){
					res[pos] = pos-1;
				} else {
					res[pos] = pos;
				}
			} else {
				res[pos] = 0;
			}
		}
	}
}

/**
 * Reads the bitmap and creates the equivalences found in coulmns
 */
__global__ void col_equivalences( int *input, int *res, int w, int h ){

    int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	//The position is increased by one, so the position 0 is reserved to the background value and it's easier to access the other values later.
    int pos = (j + i * w) + 1;
	int upper = j + (i - 1) * w;

	if ( i == 0 && j == 0 ){
		res[0] = 0;
	}

	if ( i < h && j < w ){
		if ( i == 0 ){
			res[pos] = pos * input[pos - 1];
		} else {
			if ( input[pos - 1] ){
				if ( input[upper] ){
					res[pos] = pos-(w);
				} else {
					res[pos] = pos;
				}
			} else {
				res[pos] = 0;
			}
		}
	}
}

/**
 * Sets the labels of the result from the vector labels
 */
__global__ void set_labels( int *input, int *result, int *labels, int w, int h ){
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	//The position is increased by one, so the position 0 is reserved to the background value and it's easier to access the other values later.
    int pos = (j + i * w) + 1;
	int root = labels[pos];

		while ( root != labels[root] ){
			root = labels[root];
		}
		labels[pos] = root;
		result[pos - 1] = root;
		
}

/**
 * Merging column values and row values, getting the provvisional label values
 */
__global__ void merge( int *input, int *result, int *row, int *col, int *label, int w, int h){

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	//The position is increased by one, so the position 0 is reserved to the background value and it's easier to access the other values later.
    int pos = (j + i * w) + 1;
	if ( i < h && j < w ){
		label[pos] = 0;
		if ( input[pos - 1] ){
			if ( col[pos] == pos && row[pos] == pos ){
				label[pos] = pos;
			} else if ( col[pos] && col[pos] != pos){
				label[pos] = col[pos];
			} else if ( row[pos] && row[pos] != pos ){
				label[pos] = row[pos];
			}
		} else {
			result[pos - 1] = 0;
		}
	}
}

/**
 * Checks if root leads to test_root
 */
int check_cycles( int *labels, int test_root, int root ){

	while ( labels[root] != root && labels[root] != test_root ){
		root = labels[root];
	}

	return labels[root] != test_root;
}

/**
 * Checks the roots and updates if no cycles are found
 */
void update_label( int root_to_update, int root_to_set, int *labels ){
	int is_acyclic;

	//Searching the root of the label to update
	while ( root_to_update != labels[root_to_update] && root_to_set != root_to_update ){
		root_to_update = labels[root_to_update];
	}

	is_acyclic = check_cycles( labels, root_to_update, root_to_set );
	if ( labels[root_to_set] != root_to_update && root_to_update != root_to_set && is_acyclic){
		labels[root_to_update] = root_to_set;
	}
}

/**
 * Creates the result matrix with labels from an initial binary matrix in connectivity 4
 */
void label( bitmap *bm ){
	int i, j;
	int pos;
	int upper;
	int left;
	int upper_left;
	int root, my_root;
	int size;

	int *label;

	int *d_input, *d_result;
	int *d_row, *d_col, *d_label;

	dim3 block( BLKDIM, BLKDIM );
	dim3 grid( (bm->w + BLKDIM - 1)/BLKDIM, (bm->h + BLKDIM - 1)/BLKDIM );

	size = bm->h * bm->w;

	size_t byte_size = size * sizeof(int);
	size_t label_size = (size + 1) * sizeof(int);

	label = (int*)malloc( (size + 1) * sizeof(int) );
	
	//Input and result contain the matrix as it is while row, col and label have an offset to contain the background value (0) at position 0
	cudaSafeCall( hipMalloc((int**) &d_input, byte_size) );
	cudaSafeCall( hipMalloc((int**) &d_result, byte_size) );
	//The size is increased by 1, so position 0 is reserved to the 0 value (background) and all the values are easily reached later
	cudaSafeCall( hipMalloc((int**) &d_row, label_size) );
	cudaSafeCall( hipMalloc((int**) &d_col, label_size) );
	cudaSafeCall( hipMalloc((int**) &d_label, label_size) );

	cudaSafeCall( hipMemcpy(d_input, bm->image, byte_size, hipMemcpyHostToDevice) );

	row_equivalences<<<grid, block>>>(d_input, d_row, bm->w, bm->h);
	cudaCheckError();
	col_equivalences<<<grid, block>>>(d_input, d_col, bm->w, bm->h);
	cudaCheckError();

	//Merging column values and row values, getting the provvisional label values
	merge<<<grid, block>>>(d_input, d_result, d_row, d_col, d_label, bm->w, bm->h);
	cudaCheckError();

	cudaSafeCall( hipFree( d_row ) );
	cudaSafeCall( hipFree( d_col ) );

	//Setting provisional labels
	set_labels<<<grid, block>>>(d_input, d_result, d_label, bm->w, bm->h);
	cudaCheckError();

	cudaSafeCall( hipMemcpy( label, d_label, label_size, hipMemcpyDeviceToHost ) );

	/**
	 * Refining of the local equivalences
	 * In particular removing the equivalences of the following type
	 * 
	 * 			* * * * *
	 *			* * a * *
	 *			* b a * *
	 *			* * * * *
	 * 
	 * This is the only type of the equivalences that could be found after the previous passes of the algorithm.
	 */
	for (i = 1; i < bm->h; i++){
		for (j = 1; j < bm->w; j++){
			pos = (j + i * bm->w) + 1;
			upper = j + (i-1) * bm->w;
			left = (j - 1) + i * bm->w;
			upper_left = (j-1) + (i-1) * bm->w;
			if ( bm->image[pos - 1] ){
				my_root = label[pos];
				if ( bm->image[upper] && bm->image[left] && !bm->image[upper_left] ){
					root = label[pos-1];
					update_label(my_root, root, label);
				}
			}
		}
	}

	cudaSafeCall( hipMemcpy( d_label, label, label_size, hipMemcpyHostToDevice ) );
	
	//Setting final labels
	set_labels<<<grid, block>>>(d_input, d_result, d_label, bm->w, bm->h);
	cudaCheckError();

	cudaSafeCall( hipMemcpy(bm->result, d_result, byte_size, hipMemcpyDeviceToHost) );

	free(label);
	cudaSafeCall( hipFree( d_label ) );
}

/**
 * Free the bitmap data structure
 */
void free_bitmap( bitmap *bm )
{
	free( bm->image );
	free( bm->result );
}

int main( void )
{
    bitmap bm;
    double tstart, elapsed;
	int x = 0, y = 0;
	int pos = 0, left = 0, upper = 0;
	int val = 0;
	int isCorrect = 1;
    
    read_input(stdin, &bm);

	tstart = hpc_gettime();
	label(&bm);
	elapsed = hpc_gettime() - tstart;

    fprintf(stderr, "Elapsed time = %f sec\n", elapsed);
	
	//Printing the result
	printf("Result:\n");
    for ( int i = 0; i < bm.h; i++ ){
		for ( int j = 0; j < bm.w; j++ ){
			pos = j + i * bm.w;
			left = (j - 1) + i * bm.w;
			upper = j + (i - 1) * bm.w;
			printf("%9d ", bm.result[pos]);
			if ( i && j ){
				//Checking if the result provided by the algorithm is correct
				if ( bm.image[pos] ){
					if ( (bm.image[pos] == 1 && bm.result[pos] == 0) || (bm.image[pos] == 0 && bm.result[pos] != 0) ){
						x = j;
						y = i;
						val = bm.result[pos];
						isCorrect = 0;
					}
					if ( bm.result[left] && bm.result[left] != bm.result[pos]){
						x = j;
						y = i;
						val = bm.result[pos];
						isCorrect = 0;
					}
					if ( bm.result[upper] && bm.result[upper] != bm.result[pos]){
						x = j;
						y = i;
						val = bm.result[pos];
						isCorrect = 0;
					}
				}
			}
		}
		printf("\n");
	}

	if ( isCorrect ){
		fprintf(stderr, "Correct\n");
	} else {
		fprintf(stderr, "Result Wrong. Last wrong value found: X = %d, Y = %d, val = %d\n", x, y, val);
	}

    free_bitmap(&bm);
    return EXIT_SUCCESS;    
}
